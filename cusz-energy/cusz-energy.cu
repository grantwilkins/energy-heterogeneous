/**
 * @file capi.cu
 * @author Jiannan Tian
 * @brief
 * @version 0.3
 * @date 2022-05-06
 *
 * (C) 2022 by Washington State University, Argonne National Laboratory
 *
 */

#include <thread>
#include <cstdlib>
#include <string>
#include "../cusz-latest/include/cusz.h"
#include "../cusz-latest/include/cuszapi.hh"

#include "cli/quality_viewer.hh"
#include "cli/timerecord_viewer.hh"
#include <assert.h>
extern "C" {
    #include "papi.h"
}

template <typename T>
void f(std::string fname)
{
    int dev {};
    hipGetDevice(&dev);
    hipSetDevice(dev);
    std::string const filename = {"stats.csv"};
    nvmlClass nvml(dev, filename);
    
    /* For demo, we use 3600x1800 CESM data. */
    auto len = 3600 * 1800;

    cusz_header header;
    uint8_t*    exposed_compressed;
    uint8_t*    compressed;
    size_t      compressed_len;

    T *d_uncompressed, *h_uncompressed;
    T *d_decompressed, *h_decompressed;

    /* cuSZ requires a 3% overhead on device (not required on host). */
    size_t uncompressed_memlen = len * 1.03;
    size_t decompressed_memlen = uncompressed_memlen;

    /* code snippet for looking at the device array easily */
    auto peek_devdata = [](T* d_arr, size_t num = 20) {
        thrust::for_each(thrust::device, d_arr, d_arr + num, [=] __device__ __host__(const T i) { printf("%f\t", i); });
        printf("\n");
    };

    // clang-format off
    hipMalloc(     &d_uncompressed, sizeof(T) * uncompressed_memlen );
    hipHostMalloc( &h_uncompressed, sizeof(T) * len );
    hipMalloc(     &d_decompressed, sizeof(T) * decompressed_memlen );
    hipHostMalloc( &h_decompressed, sizeof(T) * len );
    // clang-format on

    /* User handles loading from filesystem & transferring to device. */
    io::read_binary_to_array(fname, h_uncompressed, len);
    hipMemcpy(d_uncompressed, h_uncompressed, sizeof(T) * len, hipMemcpyHostToDevice);

    /* a casual peek */
    printf("peeking uncompressed data, 20 elements\n");
    peek_devdata(d_uncompressed, 20);

    hipStream_t stream;
    hipStreamCreate(&stream);

    // using default
    // cusz_framework* framework = cusz_default_framework();
    // alternatively
    cusz_framework fw = cusz_framework{
        .pipeline     = Auto,
        .predictor    = cusz_custom_predictor{.type = LorenzoI},
        .quantization = cusz_custom_quantization{.radius = 512},
        .codec        = cusz_custom_codec{.type = Huffman}};
    cusz_framework* framework = &fw;

    // Brace initializing a struct pointer is not supported by all host compilers
    // when nvcc forwards.
    // cusz_framework* framework = new cusz_framework{
    //     .pipeline     = Auto,
    //     .predictor    = cusz_custom_predictor{.type = LorenzoI},
    //     .quantization = cusz_custom_quantization{.radius = 512},
    //     .codec        = cusz_custom_codec{.type = Huffman}};

    cusz_compressor* comp       = cusz_create(framework, FP32);
    cusz_config*     config     = new cusz_config{.eb = 1e-1, .mode = Rel};
    cusz_len         uncomp_len = cusz_len{3600, 1800, 1, 1, 1.03};
    cusz_len         decomp_len = uncomp_len;

    std::thread threadStart(&nvmlClass::getStats, &nvml);

    cusz::TimeRecord compress_timerecord;
    cusz::TimeRecord decompress_timerecord;

    {
        cusz_compress(
            comp, config, d_uncompressed, uncomp_len, &exposed_compressed, &compressed_len, &header,
            (void*)&compress_timerecord, stream);

        /* User can interpret the collected time information in other ways. */
        cusz::TimeRecordViewer::view_compression(&compress_timerecord, len * sizeof(T), compressed_len);

        /* verify header */
        printf("header.%-*s : %x\n", 12, "(addr)", &header);
        printf("header.%-*s : %lu, %lu, %lu\n", 12, "{x,y,z}", header.x, header.y, header.z);
        printf("header.%-*s : %lu\n", 12, "filesize", ConfigHelper::get_filesize(&header));
    }

    std::thread threadKill(&nvmlClass::killThread, &nvml);
    threadStart.join();
    threadKill.join();

    /* If needed, User should perform a memcopy to transfer `exposed_compressed` before `compressor` is destroyed. */
    hipMalloc(&compressed, compressed_len);
    hipMemcpy(compressed, exposed_compressed, compressed_len, hipMemcpyDeviceToDevice);

    {
        cusz_decompress(
            comp, &header, exposed_compressed, compressed_len, d_decompressed, decomp_len,
            (void*)&decompress_timerecord, stream);

        cusz::TimeRecordViewer::view_decompression(&decompress_timerecord, len * sizeof(T));
    }

    /* demo: offline checking (de)compression quality. */
    /* load data again    */ hipMemcpy(d_uncompressed, h_uncompressed, sizeof(T) * len, hipMemcpyHostToDevice);
    /* perform evaluation */ cusz::QualityViewer::echo_metric_gpu(d_decompressed, d_uncompressed, len, compressed_len);

    cusz_release(comp);

    hipFree(compressed);
    // delete compressor;

    hipStreamDestroy(stream);
}

int main(int argc, char** argv)
{
    if (argc < 2) {
        printf("PROG /path/to/cesm-3600x1800\n");
        exit(0);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int EventSet = PAPI_NULL;
    long long *values, *values2;
    int num_events=0;
    int code;
    char event_names[MAX_powercap_EVENTS][PAPI_MAX_STR_LEN];
    char event_descrs[MAX_powercap_EVENTS][PAPI_MAX_STR_LEN];
    char units[MAX_powercap_EVENTS][PAPI_MIN_STR_LEN];
    int data_type[MAX_powercap_EVENTS];
    int r,i;
    int retval = 0;
    const PAPI_component_info_t *cmpinfo = NULL;
    PAPI_event_info_t evinfo;
    long long before_time,after_time;
    double elapsed_time;

    assert(PAPI_library_init( PAPI_VER_CURRENT ) == PAPI_VER_CURRENT);
    
    assert(PAPI_create_eventset(&EventSet) == PAPI_OK);

    assert(PAPI_add_named_event( EventSet, "nvml:::NVIDIA_GeForce_GTX_1070_Ti:device_0:power" ) == PAPI_OK);
    values= (long long *)calloc(1,sizeof( long long ) );

    assert(PAPI_start(EventSet) == PAPI_OK);

    hipEventRecord(start);
    f<float>(std::string(argv[1]));
    hipEventRecord(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    assert(PAPI_stop(EventSet, values) == PAPI_OK);

    power_W = values[0] / 1e3;
    energy_J = power_W * milliseconds * 1e3;

    printf("POWER = %lfW, ENERGY = %lfJ, TIME = %lfms", power_W, energy_J, milliseconds);
    return 0;
}
